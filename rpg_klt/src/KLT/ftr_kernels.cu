#include "hip/hip_runtime.h"
#include "obj.hpp"
#include "klt.hpp"
#include "hookup.hu"
#include "fast.hu"
#include <iostream>

__device__ int alo(){
	int x = 5;
	return x;
}

__device__ int FASTalgorithme(int x, int y,int image[], int *threshold){
	int score = 0;
	score = FASTcalculus(x,y, image, (*threshold) );
	/*
	if(get_HKpoint(FASTcalculus(x,y, image, (*threshold) ))){ // test if the consecutive brighter px condition is respected
		score = feature_score_calculus( x, y, image);
	}
	else {
		score = 0;
	}*/
	return score;
}


__global__ void kernel_feature_calculus(int *image, params_t *block_param, coor_t *ftr_final_list, int *threshold){

	
	extern __shared__ coor_t feature_list[]; // list of all the features to find the best one with a reduction after
	int t_id = threadIdx.x + threadIdx.y * blockDim.x;
	int max_score = 0;
	int new_score = 0;
	int x_px, y_px; 
	//calculus of features
	for(int line = 0; line < (*block_param).height / blockDim.y ; line ++) {
		//new score calculus
		x_px = blockIdx.x * (*block_param).width + threadIdx.x;
		y_px = blockIdx.y * (*block_param).height + threadIdx.y + line*blockDim.y;
		new_score = FASTalgorithme( x_px, y_px, image, threshold);

		__syncthreads(); 
		if (max_score < new_score){
			new_score = max_score;
			feature_list[t_id].x = x_px;
			feature_list[t_id].y = y_px;
			feature_list[t_id].score = max_score;
		}
	} 

/*
	//reduction to find the best feature within the block 
	int nb_threads_alive = blockIdx.y *(* block_param).width / 2;
	while((nb_threads_alive > 1) && (t_id <= nb_threads_alive) ){
		//__syncthreads();
		nb_threads_alive /=2;
		if ( feature_list[t_id + nb_threads_alive].score > feature_list[t_id].score){
			feature_list[t_id].x = feature_list[t_id + nb_threads_alive].x;
			feature_list[t_id].y = feature_list[t_id + nb_threads_alive].y;
			feature_list[t_id].score = feature_list[t_id + nb_threads_alive].score;
		} // if not, we already have the best feature on this position
	}
	if(t_id == 0){ // last t_id should be 0 accordind to the reduction process, it containes the best feature
		ftr_final_list[0].x = feature_list[0].x; //not finished
	        ftr_final_list[0].y = feature_list[0].y; //not finished
		ftr_final_list[0].score = feature_list[0].score; //not finished	
		//dont forget to uptade the lenght with atomic cuda operation
	}*/
	 
	return;
}

__global__ void kernel_tester(int *img, coor_t* ftr, int * thresh, params_t * blk){
	//printf("marche \n");
	extern __shared__ coor_t common[];
	ftr[0].x = (* thresh)+ (*blk).width;
	ftr[0].y = img[615*364];
	return;
}

void wrapper_kernel_feature_calculus(int image[], params_t block_param, params_t img_param, features * ftr_final_list, int threshold){

	//arg management
	int *img_device;
	hipMalloc((void **) &img_device, sizeof(int)*img_param.width*img_param.height );
	hipMemcpy(img_device, image, sizeof(int)*img_param.width*img_param.height, hipMemcpyHostToDevice);


	coor_t *lst_ftr_device;
	hipMalloc((void **) &lst_ftr_device, sizeof(coor_t)*N_MAX_FEATURE);
	//hipMemcpy(lst_ftr_device, (*ftr_final_list).list, sizeof(coor_t), hipMemcpyHostToDevice);


	int * thresh_device;
	int * thresh_host = &threshold;
	hipMalloc((void **) &thresh_device, sizeof(int));
	hipMemcpy(thresh_device, thresh_host, sizeof(int), hipMemcpyHostToDevice);

	params_t * block_device;
	params_t * block_host = &block_param;
	hipMalloc((void **) &block_device, sizeof(params_t));
	hipMemcpy(block_device, block_host,sizeof(params_t),hipMemcpyHostToDevice);

        int x_block = (int) img_param.width/block_param.width;
        int y_block = (int) img_param.height/block_param.height;
	dim3 blockDimension(x_block, y_block);
	

	dim3 threadsPerBlock(1,1);
	switch(block_param.width){
	case 256 :	{
		threadsPerBlock.x = block_param.width;
		threadsPerBlock.y = 1;
		break;
	}
	case 128 : {
                threadsPerBlock.x = block_param.width;
                threadsPerBlock.y = 2;
		break;
	}
	default: {
                threadsPerBlock.x = block_param.width;
                threadsPerBlock.y = 4;
		break;
	}

	}

	kernel_feature_calculus<<<blockDimension, threadsPerBlock, sizeof(coor_t)*threadsPerBlock.x*threadsPerBlock.y>>>(img_device, block_device, lst_ftr_device, thresh_device);
	//kernel_tester<<<blockDimension,threadsPerBlock, sizeof(coor_t)*threadsPerBlock.x*threadsPerBlock.y>>>(img_device,lst_ftr_device, thresh_device, block_device);
	//hipDeviceSynchronize();

	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
    		printf("CUDA error: %s\n", hipGetErrorString(error));
	}

	
        hipMemcpy( (*ftr_final_list).list, lst_ftr_device, sizeof(coor_t)*N_MAX_FEATURE, hipMemcpyDeviceToHost);
        std::cout << (*ftr_final_list).list[0].x << std::endl;
        std::cout << (*ftr_final_list).list[0].y << std::endl;
	hipFree(img_device);
	hipFree(lst_ftr_device);
	hipFree(thresh_device);
	hipFree(block_device); 

	return;
}

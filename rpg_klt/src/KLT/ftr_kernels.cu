#include "hip/hip_runtime.h"
#include "obj.hpp"
#include "klt.hpp"
#include "hookup.hu"
#include "fast.hu"

__device__ int FASTalgorithme(int x, int y,int image[], int threshold){
	int score = 0;
	
	if(get_HKpoint(FASTcalculus(x,y, image, threshold))){ // test if the consecutive brighter px condition is respected
		score = feature_score_calculus( x, y, image);
	}
	else {
		score = 0;
	}
	return score;
}


__global__ void kernel_feature_calculus(int image[], params_t block_param, features *ftr_final_list, int threshold){
	extern __shared__ coor_t feature_list[]; // list of all the features to find the best one with a reduction after
	
	int max_score = 0;
	int new_score = 0;
	int x_px, y_px;
	//calculus of features
	for(int line = 0; line < block_param.height / L ; line ++) {
		//new score calculus
		x_px = blockIdx.x * block_param.width + threadIdx.x;
		y_px = blockIdx.y * block_param.height + threadIdx.y * (block_param.height / L) + line;
		new_score = FASTalgorithme( x_px, y_px, image, threshold);
		__syncthreads(); 
		if (max_score < new_score){
			new_score = max_score;
			feature_list[threadIdx.x + threadIdx.y * L ].x = x_px;
			feature_list[threadIdx.x + threadIdx.y * L ].y = y_px;
			feature_list[threadIdx.x + threadIdx.y * L ].score = max_score;
		}
	}

	//reduction to find the best feature within the block 
	int t_id = threadIdx.x + threadIdx.y * L ;
	int nb_threads_alive = L * block_param.width / 2;
	while((nb_threads_alive > 1) && (t_id <= nb_threads_alive) ){
		nb_threads_alive /=2;
		if ( feature_list[t_id + nb_threads_alive].score > feature_list[t_id].score){
			feature_list[t_id].x = feature_list[t_id + nb_threads_alive].x;
			feature_list[t_id].y = feature_list[t_id + nb_threads_alive].y;
			feature_list[t_id].score = feature_list[t_id + nb_threads_alive].score;
		} // if not, we already have the best feature on this position
	}
	if(t_id == 0){ // last t_id should be 0 accordind to the reduction process, it containes the best feature
		//ftr_final_list.list[blockIdx.x ] = (*feature_list)[0]; //not finished 
		//dont forget to uptade the lenght with atomic cuda operation
	}
	return;
}

void wrapper_kernel_feature_calculus(int image[], params_t block_param, params_t img_param, features* ftr_final_list, int threshold){

	//arg management
	int *img_device;
	hipMalloc((void **) &img_device, sizeof(int)*img_param.width*img_param.height );
	hipMemcpy(img_device, image, sizeof(int)*img_param.width*img_param.height, hipMemcpyHostToDevice);
	features *ftr_device;
	hipMalloc((void **) &ftr_device, sizeof(features));
	hipMemcpy(ftr_device, ftr_final_list, sizeof(features), hipMemcpyHostToDevice);


        int x_block = (int) img_param.width/block_param.width;
        int y_block = (int) img_param.height/block_param.height;
	dim3 blockDimension(x_block, y_block);
	switch(block_param.width){
	case 256 :	
	{
		dim3 threadsPerBlock(256,1);
		kernel_feature_calculus<<<blockDimension, threadsPerBlock, block_param.width*block_param.height*sizeof(coor_t)>>>(img_device, block_param, ftr_device, threshold);
		break;
	}
	case 128 :
	{
		dim3 threadsPerBlock(128,2);
		kernel_feature_calculus<<<blockDimension, threadsPerBlock, block_param.width*block_param.height*sizeof(coor_t)>>>(img_device, block_param, ftr_device, threshold);
		break;
	}
	default:
	{
		dim3 threadsPerBlock(block_param.width,4);
		kernel_feature_calculus<<<blockDimension, threadsPerBlock, block_param.width*block_param.height*sizeof(coor_t)>>>(img_device, block_param, ftr_device, threshold);
		break;
	}
	}
	hipMemcpy(ftr_final_list, ftr_device, sizeof(int)*img_param.width*img_param.height, hipMemcpyDeviceToHost);
	hipFree(img_device);
	hipFree(ftr_device);

	return;
}

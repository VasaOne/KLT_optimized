#include "hip/hip_runtime.h"
#include <stdio.h>

#include "obj.hpp"
#include "fast.hu"

#define range_condition(x,y,element_id) (0<= x + elementFAST[element_id][0]) && (x + elementFAST[element_id][0] < WIDTH_IMAGE) && (0 <= y + elementFAST[element_id][1]) && (y + elementFAST[element_id][1] < HEIGHT_IMAGE) //macro to check if a point is inside the image

#define img_val_px(x, y, image) image[y*WIDTH_IMAGE + x]


 __device__ int elementFAST[16][2] = // {... {x,y} ...}
{
	{0,3},
	{1,3},
	{2,2},
	{3,1},
	{3,0},
	{3,-1},
	{2,-2},
	{1,-3},
	{0,-3},
	{-1,-3},
	{-2,-2},
	{-3,-1},
	{-3,0},
	{-3,1},
	{-2,2},
	{-1,3}
};


__device__ int  fastBrighter(int center_px, int orbit_px, int threshold){
	if( ( (center_px - orbit_px) >= threshold) || ( (orbit_px - center_px) >= threshold ) ){
		return 1;
	}
	return 0;
}



__device__ int  FASTcalculus(int x, int y, int image[], int threshold){
	int val = 0;
	for (int element_id = 0; element_id < 16; element_id ++){ // calculus of sum of FAST elements
		if (range_condition( x, y, element_id ) ){ // check if pixel is inside the range of the image
                        int nx = x + elementFAST[element_id][0];
                        int ny = y + elementFAST[element_id][1];			
			val = val | ( fastBrighter( img_val_px(x, y, image), img_val_px(nx, ny, image), threshold ) << element_id);
		}
	}
	return val;
}

__device__ int feature_score_calculus(int x_center, int y_center, int image[]){
	int score = 0;
	for(int element_id = 0; element_id < 16; element_id ++){
		int nx = x_center + elementFAST[element_id][0];
		int ny = y_center + elementFAST[element_id][1];
		score += abs( img_val_px(nx, ny, image) - img_val_px(x_center, y_center, image) );
	}
	return score;
	

}

/*
__global__ void testor(){
	int dk = fastBrighter(56, 68, 30);
	int br_1 = fastBrighter(56, 198,30);
	int br_2 = fastBrighter(192, 56, 30);
	printf("should be dark: %i \n", dk);
	printf("should be brighter: %i \n", br_1);
	printf("should be brighter: %i \n", br_2);
	
	int small_img[16*9] = { 0,0,0,0,115,115,115,115,115,30,75,75,0,0,0,0,
       			        0,0,0,0,150,156,154,153,150,75,76,45,0,0,0,0,
                                0,0,0,4,124,124,128,127,126,12,75,46,0,0,0,0,
                                0,0,0,0,115,115,115,115,115,30,75,75,0,0,0,0,
                                0,0,0,0,150,156,154,153,150,75,76,45,0,0,0,0,
                                0,0,0,4,12,124,128,127,126,12,75,46,0,0,0,0,
                                0,0,0,0,11,115,115,115,115,30,75,75,0,0,0,0,
                                0,0,0,0,150,156,154,153,150,75,76,45,0,0,0,0,
                                0,0,0,4,12 ,14 ,12 ,127,19 ,12,7 ,6 ,0,0,0,0	
	};
	printf("get 8 7 value: %i \n", img_val_px(8, 7, small_img));
	int fast_result = FASTcalculus(8, 4, small_img, 30);
	printf("should be I dont know: %i \n", fast_result);

	int score_ftr = feature_score_calculus(8,4,small_img);
	printf("score of the feature (8,4): %i \n", score_ftr);
	return;
}

int main(){
	testor<<<1,1>>>();
	hipDeviceSynchronize();
	return 1;
}
*/

#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hookup.hu"
#include "klt.hpp"


__device__ char HookUpTable[HK_size];

__device__ void set_HKpoint(int addr){
	//printf("full addr: %i place: %i, val: %i \n", addr,addr & 0x1FFF, 1 << (addr >> 13) );
	HookUpTable[addr & 0x1FFF] = (HookUpTable[addr & 0x1FFF] | (1 << (addr >> 13)));
}

__device__ void clear_HKpoint(int addr){
	HookUpTable[addr & 0x1FFF] = HookUpTable[addr & 0x1FFF] & (~(1 << (addr >> 13)));
}

__device__ int get_HKpoint(int addr){
	return HookUpTable[addr & 0x1FFF] & (1 << (addr >> 13));
}

__global__ void cleanHK(){
	int t_id = threadIdx.x;
	HookUpTable[t_id] = 0;
}

__device__ void addr_Init(int addr, int pointcondition){
	int count = 0; // count the maximum of consecutive 1
	for(int loop = 0; loop < 2; loop ++){ // count 2 times to include the 16 -> 1 path
		for(int it = 0; it < 16; it ++){
			if((addr & (1 << it)) >> it ){
				count ++;
				if (count == pointcondition){
					set_HKpoint(addr);
					return;
				}
			} 
			else {
				count = 0;
			}
		}
	}
	clear_HKpoint(addr);
}

__global__ void HookUpInit(int pointcondition){
	for(int addr = 0; addr< 8192; addr++){
		int new_addr = 0;
		for (int i =0; i < 8; i++){
			new_addr = addr | ( i << 13 );
			addr_Init(new_addr, pointcondition);
		}
	}
}
__global__ void HookUpTester(int pointcondition){
        int addr = threadIdx.x;
        int new_addr = 0;
        for (int i =0; i < 256; i++){
                new_addr = addr | ( i << 8 );
                if (get_HKpoint(new_addr)){
			printf("addr: %i \n", new_addr);
		}
        }
}


void wrapper_kernel_HKinit(int pointcondition){
	cleanHK<<<256,256>>>();
	HookUpInit<<<1,1>>>(pointcondition);
	//HookUpTester<<<1,256>>>(pointcondition);
	return;
}

/*
__global__ void testor(){
	printf("for 3: %i \n", get_HKpoint(3));
	printf("for 5: %i \n", get_HKpoint(5));
	printf("for 15: %i \n", get_HKpoint(15));
	printf("for jsp mais ca doit faire faux %i \n", get_HKpoint(0b11101110) );

}
int main(){
	printf("expected: 4");
	HookUpInit<<<32,256>>>(4);
	testor<<<1,1>>>();
	hipDeviceSynchronize();
	return 1;
}
*/

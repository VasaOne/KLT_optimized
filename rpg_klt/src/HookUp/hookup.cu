#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hookup.hu"




__device__ void set_HKpoint(int addr){
	HookUpTable[addr & 0x1FF] = HookUpTable[addr & 0x1FF] | (1 << (addr >> 13));
}

__device__ void clear_HKpoint(int addr){
	HookUpTable[addr & 0x1FF] = HookUpTable[addr & 0x1FF] & (~(1 << (addr >> 13)));
}

__device__ int get_HKpoint(int addr){
	return HookUpTable[addr & 0x1FF] & (1 << (addr >> 13));
}

__global__ void HookUpInit(int pointcondition){
	int addr = blockIdx.x * blockDim.x + threadIdx.x;
	int count = 0; // count the maximum of consecutive 1
	for(int loop = 0; loop < 2; loop ++){ // count 2 times to include the 16 -> 1 path
		for(int it = 0; it < 16; it ++){
			if((addr & (1 << it)) >> it ){
				count ++;
				if (count == pointcondition){
					set_HKpoint(addr);
					return;
				}
			} 
			else {
				count = 0;
			}
		}
	}
	clear_HKpoint(addr);
}
__global__ void testor(){
	printf("for 3: %i \n", get_HKpoint(3));
	printf("for 5: %i \n", get_HKpoint(5));
	printf("for 15: %i \n", get_HKpoint(15));
	printf("for jsp mais ca doit faire faux %i \n", get_HKpoint(0b11101110) );

}
int main(){
	printf("expected: 4");
	HookUpInit<<<32,256>>>(4);
	testor<<<1,1>>>();
	hipDeviceSynchronize();
	return 1;
}
